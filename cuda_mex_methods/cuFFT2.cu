#include "hip/hip_runtime.h"
/*

version of psf2otf for m = n

*/
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"
#include "hip/hip_runtime_api.h"
#include <hipfft/hipfft.h>

template <typename T> __global__ void copyRealFromComplex(hipfftDoubleComplex* C, T* R, int n);
void check_cufft(hipfftResult status);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

	 /* Declare all variables.*/
    mxGPUArray *A, *B;
    double *d_A, *d_B;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs < 2) || !(mxIsGPUArray(prhs[0]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    A = mxGPUCopyFromMxArray (prhs[0]); // mxGPUCreateFromMxArray(prhs[0]);
	int N = mxGetScalar(prhs[1]);

    /*
     * Verify that A really is a double array before extracting the pointer.
     */
    if ((mxGPUGetClassID(A) != mxDOUBLE_CLASS)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_A = (double *)(mxGPUGetData(A));

    /* Create a GPUArray to hold the result and get its underlying pointer. */

    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_INITIALIZE_VALUES );
    d_B = (double *)(mxGPUGetData(B));
	// ----------------- END OF MEX FILE STARTING CONFIGURATION ----------------------------

	hipfftHandle plan2d;
	hipfftDoubleComplex *output;
	int n = N*(N/2+1);
	checkCudaErrors(hipMalloc((void**)&output, sizeof(hipfftDoubleComplex)*n));


	// cufft plan initialization
	if (hipfftPlan2d(&plan2d, N, N, HIPFFT_D2Z) != HIPFFT_SUCCESS){	// N - number of input samples (length of input data)
		hipFree(output);
		mexErrMsgIdAndTxt(errId, "plan initialization failed, cufft error code\n");
	}

	// exec
	if (hipfftExecD2Z(plan2d, d_A, output) != HIPFFT_SUCCESS){
		hipFree(output);
		hipfftDestroy(plan2d);		
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}


	int threads = N;
	int blocks = N;
	copyRealFromComplex<<<threads, blocks>>>(output, d_B, n);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());		

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);

    mxGPUDestroyGPUArray(A);
	mxGPUDestroyGPUArray(B);

	hipfftDestroy(plan2d);
	hipFree(output);
}

template <typename T> __global__ void copyRealFromComplex(hipfftDoubleComplex* C, T* R, int n){

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n){
		R[index] = C[index].x;
	}
}

void check_cufft(hipfftResult status){
	
	char const * const errId = "parallel:gpu:mexGPUExample:CufftError";
	if (status != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft error code %d\n", status);
	}
}