#include "hip/hip_runtime.h"
/*

version of psf2otf for m = n

*/
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"
#include "hip/hip_runtime_api.h"
#include <hipfft/hipfft.h>

template <typename T> __global__ void copyRealFromComplex(hipfftDoubleComplex* C, T* R, int n);
void check_cufft(hipfftResult status);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

    mxGPUArray *A, *B;
    double *d_A, *d_B;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    mxInitGPU();

    if ((nrhs < 2) || !(mxIsGPUArray(prhs[0]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    A = mxGPUCopyFromMxArray (prhs[0]); // mxGPUCreateFromMxArray(prhs[0]);
	int N = mxGetScalar(prhs[1]);

    if ((mxGPUGetClassID(A) != mxDOUBLE_CLASS)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    d_A = (double *)(mxGPUGetData(A));

    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_INITIALIZE_VALUES );
    d_B = (double *)(mxGPUGetData(B));
	// ----------------- END OF MEX FILE STARTING CONFIGURATION ----------------------------

	hipfftHandle plan, plan_inverse;
	hipfftDoubleComplex *output;
	int n = (N/2)+1;
	checkCudaErrors(hipMalloc((void**)&output, sizeof(hipfftDoubleComplex)*n));


	// cufft plan initialization
	if (hipfftPlan1d(&plan, N, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS){	// N - number of input samples (length of input data)
		hipFree(output);
		mexErrMsgIdAndTxt(errId, "plan initialization failed, cufft error code\n");
	}

	if (hipfftPlan1d(&plan_inverse, N, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "inverse plan initialization failed, cufft error code\n");
	}

	// exec
	if (hipfftExecD2Z(plan, d_A, output) != HIPFFT_SUCCESS){
		hipFree(output);
		hipfftDestroy(plan);		
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}

	
	if (hipfftExecZ2D(plan_inverse, output, d_B) != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}


	int threads = N;
	int blocks = N;
	//copyRealFromComplex<<<threads, blocks>>>(output, d_B, n);
	//hipDeviceSynchronize();
	//checkCudaErrors(hipGetLastError());		

    plhs[0] = mxGPUCreateMxArrayOnGPU(B);

    mxGPUDestroyGPUArray(A);
	mxGPUDestroyGPUArray(B);

	hipfftDestroy(plan);
	hipFree(output);
}

template <typename T> __global__ void copyRealFromComplex(hipfftDoubleComplex* C, T* R, int n){

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n){
		R[index] = C[index].x;
	}
}

void check_cufft(hipfftResult status){
	
	char const * const errId = "parallel:gpu:mexGPUExample:CufftError";
	if (status != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft error code %d\n", status);
	}
}