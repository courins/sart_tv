#include "hip/hip_runtime.h"
/*

version of psf2otf for m = n

*/
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"
#include "hip/hip_runtime_api.h"
#include <hipfft/hipfft.h>

template<typename T> __global__ void psf_from_fft(hipfftDoubleComplex* v, int N, int n, T* real_fft, T* result);
template<typename T> __global__ void add_and_divide_cut_complex(T * Numer1, cufftDoubleCompex* fft2_rhs, T* Denom, int fft2_rows, int fft2_cols);

void check_cufft(hipfftResult status);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

	 /* Declare all variables.*/
    mxGPUArray *Numer1, *rhs, *Denom, *U;
    double *d_Numer1, *d_rhs, *d_Denom, *d_U;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs < 4) || !(mxIsGPUArray(prhs[0])) || !(mxIsGPUArray(prhs[1])) || !(mxIsGPUArray(prhs[2]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    Numer1 = mxGPUCopyFromMxArray (prhs[0]); // mxGPUCreateFromMxArray(prhs[0]);
	rhs = mxGPUCopyFromMxArray (prhs[1]);
	Denom = mxGPUCopyFromMxArray (prhs[2]);
	int N = mxGetScalar(prhs[3]);

    /*
     * Verify that A really is a double array before extracting the pointer.
     */
    if ((mxGPUGetClassID(Numer1) != mxDOUBLE_CLASS) ||(mxGPUGetClassID(rhs) != mxDOUBLE_CLASS) || (mxGPUGetClassID(Denom) != mxDOUBLE_CLASS)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_Numer1 = (double *)(mxGPUGetData(Numer1));
	d_rhs = (double *)(mxGPUGetData(rhs));
	d_Denom = (double *)(mxGPUGetData(Denom));

    /* Create a GPUArray to hold the result and get its underlying pointer. */

    U = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(Numer1),
                            mxGPUGetDimensions(Numer1),
                            mxGPUGetClassID(Numer1),
                            mxGPUGetComplexity(Numer1),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_U = (double *)(mxGPUGetData(U));
	// ----------------- END OF MEX FILE STARTING CONFIGURATION ----------------------------

	//  --------------- REC BREGMAN METHODS STARTS HERE -------------------------------------------

	
	hipfftHandle plan;


	hipfftDoubleComplex *output;
	double prd[2] = {0.57, -0.57};
	double * input, *temp_worksapce;

	int n = (N/2)+1;

	//---------------------- MEMORY ALLOCATION
	checkCudaErrors(hipMalloc((void**)&output, sizeof(hipfftDoubleComplex)*n));
	hipMalloc((void**)&input, sizeof(double)*N);
	hipMemset(input, 0, sizeof(double)*N);
	hipMalloc((void**)&temp_worksapce, sizeof(double)*N);

	hipMemcpy(input, &prd[0], sizeof(double)*2, hipMemcpyHostToDevice);


	// cufft plan initialization
	if (hipfftPlan1d(&plan, N, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS){	// N - number of input samples (length of input data)
		hipFree(output);
		mexErrMsgIdAndTxt(errId, "plan initialization failed, cufft error code\n");
	}

	// exec
	if (hipfftExecD2Z(plan, input, output) != HIPFFT_SUCCESS){
		hipFree(output);
		hipfftDestroy(plan);		
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}


	int threads = N;
	int blocks = N;
	int shared_mem_size = N*sizeof(double);
	psf_from_fft<<<threads, blocks, shared_mem_size>>>(output, N, n, temp_worksapce, d_U);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());		

	// -------------------------------------- U = IFFT2((Numer1 + FFT2(rhs))./Denom) --------------------------------------------

	// --------------------------------------- CUFFT PLAN INITIALIZATION -------------------------------

	hipfftHandle plan2d, plan_ifft;
	int fft2_output_size = N*(N/2+1);
	hipfftDoubleComplex* fft2_rhs;
	hipMalloc((void**)&fft2_rhs, sizeof(hipfftDoubleComplex)*fft2_output_size);

	if(cufftPlan2D(&plan2d, N, N, HIPFFT_D2Z) != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "plan2d initialization failed, cufft error code\n");
	}
	if(cufftPlan2D(&plan_ifft, N, N, HIPFFT_Z2D) != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "ifft plan initialization failed, cufft error code\n");
	}

	// --------------------------------------- FFT2 EXECUTION -------------------------------
	// ftt(rhs)
	if(hipfftExecD2Z(plan2d, d_rhs, fft2_rhs) != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}

	// ------------------------------------- (Numer1 + fft2(rhs))./Denom
	threads = N;
	blocks = N/2+1;
	add_and_divide_cut_complex<<<threads, block>>>(d_Numer1, fft2_rhs, d_Denom, N, N/2+1);

	// inverse fft ifft2((Numer1 + fft2(rhs))./Denom)
	if(hipfftExecZ2D(plan_ifft, fft2_rhs, d_U) != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}

	// END OF MY CODE

	/* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(U);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(Numer1);
	mxGPUDestroyGPUArray(rhs);
	mxGPUDestroyGPUArray(Denom);
    mxGPUDestroyGPUArray(U);

	hipfftDestroy(plan);

	hipFree(output);
	hipFree(input);
	hipFree(temp_worksapce);
	hipFree(fft2_output_data);
}

template<typename T>
__global__ void psf_from_fft(hipfftDoubleComplex* v, int N, int n, T* real_fft, T* result){

	// blocks are rows which cover the whole length, usually the matrix is of size 512x512, the NVIDIA GPU's allows maximum number
	// of threads per block of 1024
	
	if (threadIdx.x >= n && threadIdx.x < N){
		v[threadIdx.x] = v[N-threadIdx.x];
	}

	if (threadIdx.x < N){
		hipfftDoubleComplex a = v[threadIdx.x];
		real_fft[threadIdx.x] = pow(a.x, 2.0) + pow(a.y, 2.0);
	}

	if (blockIdx.x < N && threadIdx.x < N){
		extern __shared__ double v_shared[];
		v_shared[threadIdx.x] = real_fft[threadIdx.x];

		result[threadIdx.x+blockIdx.x*N] = v_shared[threadIdx.x] + v_shared[blockIdx.x];
	}
}

template<typename T>
__global__ void add_and_divide_cut_complex(T * Numer1, cufftDoubleCompex* fft2_rhs, T* Denom, int fft2_rows, int fft2_cols){

	// IMPORTANT: Numer1 and Denom are matrices of size rows x cols, but fft2_rhs was computed by cufft in R2C plan so its total length is N*(N/2+1);
	 // operation: (Numer1 + fft2(rhs))./Denom
/*
Similar to the one-dimensional case, the frequency domain representation of real-valued
input data satisfies Hermitian symmetry, defined as: X(n1, n2, ..., nd) = X*(N1-n1,N2-n2,...,Nd-nd)
for two dimensional fft, i.e. fft2 on NxM matrix indexing is the following: X(n,m) = X*(N-n, M-m);
the length of fft2 done by cufft from NxM is: N*(M/2+1);

kernel run configuration should be fitted to this size N*(M/2+1)

*/

	if (threadIdx.x < fft2_rows && blockIdx.x < fft2_cols){

		int index = threadIdx.x + blockIdx.x*fft2_cols;
		cufftDoubleCompex t = fft_rhs[index];
		t.x += Numer1[index];
		
		// UWAGA CO JESLI JEST DZIELENIE PRZEZ ZERO??
		double divider = Denom[index];
		if (divider != 0.0){
			t.x /= divider;
			t.y /= divider;
			fft_rhs[index] = t;
		}
	}
}

void check_cufft(hipfftResult status){
	
	char const * const errId = "parallel:gpu:mexGPUExample:CufftError";
	if (status != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft error code %d\n", status);
	}
}