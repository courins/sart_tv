#include "hip/hip_runtime.h"

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"


template<typename T> __global__ void compute_wx_wy(T* Ux, T* Uy, T* bx, T* by, T* Wx, T* Wy, int rows, int cols, double tau);
template<typename T> __global__ void compute_rhs_DxtU_DytU_column_mayor_order(T* bx, T* by, T* Wx, T* Wy, T* RHS, int rows, int cols, double tau);
template<typename T> __global__ void compute_rhs_DxtU_DytU_row_mayor_order(T* bx, T* by, T* Wx, T* Wy, T* RHS, int rows, int cols, double tau);
template<typename T> __global__ void compute_Ux_Uy_column_major_order(T* U, T* Ux, T* Uy, int rows, int cols);
template<typename T> __global__ void compute_Ux_Uy_row_major_order(T* U, T* Ux, T* Uy, int rows, int cols);
template<typename T> __global__ void bregman_update(T* b, T* U, T* W, int rows, int cols, T gamma);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

	 /* Declare all variables.*/
    mxGPUArray *RHS, *Wx, *Wy, *bx, *by;
    double *d_RHS, *d_Wx, *d_Wy, *d_bx, *d_by;

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs < 6) || !(mxIsGPUArray(prhs[0])) || !(mxIsGPUArray(prhs[1])) || !(mxIsGPUArray(prhs[2])) || !(mxIsGPUArray(prhs[3]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    Wx = mxGPUCopyFromMxArray(prhs[0]);
	Wy = mxGPUCopyFromMxArray(prhs[1]);
	bx = mxGPUCopyFromMxArray(prhs[2]);
	by = mxGPUCopyFromMxArray(prhs[3]);
	int N = mxGetScalar(prhs[4]);
	double tau = mxGetScalar(prhs[5]);

    /*
     * Verify that Wx, Wy, bx, by really are double array before extracting the pointer.
     */
    if ((mxGPUGetClassID(Wx) != mxDOUBLE_CLASS) || (mxGPUGetClassID(Wy) != mxDOUBLE_CLASS) || (mxGPUGetClassID(bx) != mxDOUBLE_CLASS) || (mxGPUGetClassID(by) != mxDOUBLE_CLASS)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_Wx = (double *)(mxGPUGetData(Wx));
	d_Wy = (double *)(mxGPUGetData(Wy));
	d_bx = (double *)(mxGPUGetData(bx));
	d_by = (double *)(mxGPUGetData(by));

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    RHS = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(Wx),
                            mxGPUGetDimensions(Wx),
                            mxGPUGetClassID(Wx),
                            mxGPUGetComplexity(Wx),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_RHS = (double *)(mxGPUGetData(RHS));

	// YOUR CODE HERE
	hipError_t status;
	int threads = N;
	int blocks = N;
	//compute_rhs_DxtU_DytU_column_mayor_order<<<threads, blocks>>>(d_bx, d_by, d_Wx, d_Wy, d_RHS, N, N, tau);
	compute_rhs_DxtU_DytU_row_mayor_order<<<threads, blocks>>>(d_bx, d_by, d_Wx, d_Wy, d_RHS, N, N, tau);
	hipDeviceSynchronize();
	status = hipGetLastError();
	if(status != hipSuccess){
		mexErrMsgIdAndTxt(errId, "cuda error code %d\n", status);
	}
	// END OF YOUR CODE

	/* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(RHS);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(Wx);
	mxGPUDestroyGPUArray(Wy);
	mxGPUDestroyGPUArray(bx);
	mxGPUDestroyGPUArray(by);
    mxGPUDestroyGPUArray(RHS);

}

//version for real numbers
/*
	Compute_Wx_Wy
	
	it's very simple

	xr = Ux[i] + bx[i]
	yr = Uy[i] + by[i]
	Vr = sqrt(xr*xr + yr*yr)
	
	if ...

*/
template<typename T>
__global__ void compute_wx_wy(T* Ux, T* Uy, T* bx, T* by, T* Wx, T* Wy, int rows, int cols, double tau){
	 

	// tutaj trzeba dodac wykrywanie konfiguracji kernela tzn. siatk� blok�w i w�tk�w, wymiary blok�w
	if (blockDim.x == 1){

	}
	else if(blockDim.y == 1){

	}
	else{

	}

	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = x + blockIdx.x * cols;

	if (x < cols && y < rows){

		T xr = Ux[index] + bx[index];
		T yr = Uy[index] + by[index];
		xr = pow(xr, 2.0);
		yr = pow(yr, 2.0);
		T vr = sqrt(xr+yr);
		if (vr <= tau)
            {
                Wx[index] = 0; Wy[index] = 0;
            }
            else
            {
                vr = (vr - tau) / vr;
                Wx[index] = xr*vr; Wy[index] = yr*vr;
            }
	}

}

//version for real numbers
template<typename T>
__global__ void compute_rhs_DxtU_DytU_column_mayor_order(T* bx, T* by, T* Wx, T* Wy, T* RHS, int rows, int cols, double tau){
	 
	/*
	it is assumed that data in every matrix (bx, by, Wx, Wy, RHS) are in column major order, which is typicall for matlab

	kernel configuration: blocks are spanned to cover columns i.e. every block is one-dimensional and may visualized as a column of matrix
	
	*/

	int index = threadIdx.x+blockIdx.x*rows;

	int colt = rows*(cols-1);
    int rowt = rows-1;

	// predicates depend on block configuration
	// predicate1 - if a thread belogns to the first column (0 indexed)
	// predicate2 - if a thread (cell) is a first thread in a column (0 indexed)
	int index1 = index + (colt * (blockIdx.x == 0)) - (rows*(blockIdx.x != 0));
	int index2 = index + (rowt * (threadIdx.x == 0)) - (1*(threadIdx.x != 0));

	if (blockIdx.x < cols && threadIdx.x < rows){

		RHS[index] = tau*(bx[index] - bx[index1] - Wx[index] + Wx[index1] + by[index] - by[index2] - Wy[index] + Wy[index2]);
	}
}

template<typename T>
__global__ void compute_rhs_DxtU_DytU_row_mayor_order(T* bx, T* by, T* Wx, T* Wy, T* RHS, int rows, int cols, double tau){

	/*
	it is assumed that data in every matrix (bx, by, Wx, Wy, RHS) are in row major order, which is NOT typicall for matlab

	*/
	 
	int index = threadIdx.x+blockIdx.x*cols;

	int colt = rows*(cols-1);

	// predicates depend on block configuration
	int index1 = index + ((cols-1) * (threadIdx.x == 0)) - (1*(threadIdx.x != 0));
	int index2 = index + (colt * (blockIdx.x == 0)) - (cols*(blockIdx.x != 0));

	if (threadIdx.x < cols && blockIdx.x < rows){

		RHS[index] = tau*(bx[index] - bx[index1] - Wx[index] + Wx[index1] + by[index] - by[index2] - Wy[index] + Wy[index2]);
	}
}

template<typename T>
__global__ void compute_Ux_Uy_column_major_order(T* U, T* Ux, T* Uy, int rows, int cols){


	// shuffle instructions ??
	int index = threadIdx.x+blockIdx.x*rows;

	int ux_index = (index + rows) % (rows*cols);
	int uy_index = index + 1 - (rows*(threadIdx.x == (cols-1)));

	if (threadIdx.x < cols && blockIdx.x < rows){

		T u = U[index];
		Ux[index] = U[ux_index] -u;
		Uy[index] = U[uy_index] -u;
	}
}

template<typename T>
__global__ void compute_Ux_Uy_row_major_order(T* U, T* Ux, T* Uy, int rows, int cols){

	// mo�e t� funkcj� rozbi� na dwa kernele, obliczaj�ce Ux i Uy i wtedy mo�na by uzyska� memory access coalesced albo shuffle instructions
	//shuffle instructions ??
	int index = threadIdx.x+blockIdx.x*cols;

	// TODO przekopiowane rozwi�zanie z g�ry, czy w wersje column i row powinny si� zamienia� rows z cols ??? chyba nie!!!
	int ux_index = (index + 1) % cols;
	int uy_index = (index + cols) % (rows*cols);

	if (threadIdx.x < rows && blockIdx.x < cols){

		T u = U[index];
		Ux[index] = U[ux_index] -u;
		Uy[index] = U[uy_index] -u;
	}
}

template<typename T>
__global__ void bregman_update(T* b, T* U, T* W, int rows, int cols, T gamma){

	int index = threadIdx.x + blockIdx.x*blockDim.x + threadIdx.y*cols;

	if (index < (cols*rows)){
		b[index] = gamma*(U[index]-W[index]);
	}
}

/*
	Compute_Ux_Uy

	matrices Ux and Uy are computed independently
	
	IMPORTANT !!: this computational procedures are explained assuming that data are stored in a column major order (specific for Matlab)

	computations for Ux:	Ux[i] = U[i+rows] - U[i], basically U[i+rows] is a neighboor element from next COLUMNS (but the same row) e.g. for Aij it is Ai(j+1),
                                                      except for the last column, in this case we take the first column

	computations for Uy:    Uy[i] = U[i+1] - U[i], so taking into account matlab store order U[i+1] is the next element from the same column,
												   e.g. for Aij it is A(i+1)j, for the last element of the column, the first element is taken

*/


/*

	Compute rhs

	computations for Wx, bx: Wx[i-rows]-bx[i-rows], for element Aij take elements Ai(j-1), for the first column take apprioprate elements from the last columns
	computations for Wy, by: Wy[i-1]-by[i-1], for element Aij take elements A(i-1)j (a neighbour up), for the elements in the first row, take the last element from the column

*/