#include "hip/hip_runtime.h"
﻿#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/hip_runtime_api.h"
#include "hipfft/hipfft.h"
#include <math.h>
#include <string.h>
#include "sart_constants.h"

// cublas indexing macro
//#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//#define IN_ARGS_NUM 8

const hipsparseDirection_t dirA_col = HIPSPARSE_DIRECTION_COLUMN;
const hipsparseDirection_t dirA_row = HIPSPARSE_DIRECTION_ROW;
const hipsparseOperation_t NON_TRANS = HIPSPARSE_OPERATION_NON_TRANSPOSE;
const hipsparseOperation_t TRANS = HIPSPARSE_OPERATION_TRANSPOSE;

// JAKAŚ METODA CLEANUP BY SIĘ PRZYDAŁA DO PONIŻSZYCH METOD
static const char *_cusparseGetErrorEnum(hipsparseStatus_t status)
{
    switch (status)
    {
        case HIPSPARSE_STATUS_SUCCESS:
            return "cusparse_success";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "cusparseNotInitialized";

		case HIPSPARSE_STATUS_ALLOC_FAILED:
			return "cusparseAllocFailed";

		case HIPSPARSE_STATUS_INVALID_VALUE:
			return "cusparseInvalidValue";

		case HIPSPARSE_STATUS_ARCH_MISMATCH:
			return "cusparseArchMismatch";

		case HIPSPARSE_STATUS_MAPPING_ERROR:
			return "cusparseMappingError";
		
		case HIPSPARSE_STATUS_EXECUTION_FAILED:
			return "cusparseExecutionFailed";

		case HIPSPARSE_STATUS_INTERNAL_ERROR:
			return "cusparseInternalError";

		case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
			return "cusparseMatrixTypeNotSupported";
	}

    return "<unknown>";
}

template< typename T >
void checkCusparse2(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        //fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(result), _cusparseGetErrorEnum(result), func);
        hipDeviceReset();
        // Make sure we call CUDA Device Reset before exiting
		mexErrMsgIdAndTxt("CUSPARSE ERROR ", "CUSPARSE error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), _cusparseGetErrorEnum(result), func);
    }
}

#define checkCusparseErrors(val)           checkCusparse2 ( (val), #val, __FILE__, __LINE__ )

void checkCuda();
void checkCublas(hipblasStatus_t status);
void checkCufft(hipfftResult_t status);

void verifyArguments(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]);
void verifyRetrievedPointers(mxGPUArray const *Aval, mxGPUArray const *rowInd, mxGPUArray const *colInd, mxGPUArray const *b);
void exitProgramWithErrorMessage(char *);
void initOnes(double *p, int n);

int stopping_rule(char * stoprule, int k, int kmax);

__global__ void normalizeVectorSum(double * v, int n);
__global__ void saxdotpy(double a, double * x, double *y, double n, double *z);
__global__ void elemByElem(int n, double *x, double *y, double *z);
__global__ void absComplex(hipfftDoubleComplex * idata, double *odata, int n);

//thrust::plus<double> binary_op; 
double init = 0.0;  //słabo czytelna zmienna

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

	// [X,info,restart] = sart(A,m,n,b,K)
	// [X,info,restart] = sart(A,m,n,b,K,x0)
	// mexFunction(Aval, rowInd, colInd, nnzPerRow, nnz, rows, cols, b, K)

	char * stoprule = "NO";
    bool nonneg = false;
    bool boxcon = false;
	int casel = 1;
	double lambda = 1.9;

	/*
	this function takes following arguments:

	A_val  - vector of non-zero values of matrix A           \
	rowInd - row indices of values in matrix A				  -- this three vectors fully define matrix A
	colInd - column indices of values in matrix A			 /
	nnz  - number of non-zero values (all above vectors have length of nnz)
	rows - number of rows in matrix A
	cols - number of columns in matrix A
	b - vector B
	K - number of iterations


	optionally:
	X0 - X vector with initial values

	matrix A of size m x n (m = rows, n = cols)
	vector b of length m (rows)
	vector x of length n (cols)
	rxk - size of vector b
	
	*/

	mxGPUArray const *Aval, *rowInd, *colInd, *b, *X0;
	double const *d_Aval, *d_b;
	int const* d_rowInd, *d_colInd;

	mxGPUArray *X;
	double *d_X, *d_x0;
	double *d_rxk, *d_W, *d_V, *d_Wrxk, *d_AW, *HOST_ONES;
	
	mxInitGPU();
	verifyArguments(nlhs, plhs, nrhs, prhs);

	int args_count = -1;

	/* Retrieve input arguments */
	Aval = mxGPUCreateFromMxArray(prhs[++args_count]); // 0
	rowInd = mxGPUCreateFromMxArray(prhs[++args_count]); // 1
	colInd = mxGPUCreateFromMxArray(prhs[++args_count]); // 2
	int nnz = mxGetScalar(prhs[++args_count]); // 3
	int rows = mxGetScalar(prhs[++args_count]); // 4
	int cols = mxGetScalar(prhs[++args_count]); // 5
	b = mxGPUCreateFromMxArray(prhs[++args_count]); // 6
	int K = mxGetScalar(prhs[++args_count]); // 7
	
	const int ONES_SIZE = cols*(cols>rows) + rows*(rows>cols);
	int required_args = ++args_count; // 8

	int n_sqrt = (int) sqrt(cols);
	if (n_sqrt*n_sqrt != cols)
		exitProgramWithErrorMessage("Rozmiar n (liczba kolumn) macierzy A, nie jest kwadratem liczby całkowitej");

	verifyRetrievedPointers(Aval, rowInd, colInd, b); // x0 ??
	// TODO Matlab checking error - check that the sizes of A and b match

	d_Aval = (double const *)(mxGPUGetDataReadOnly(Aval));
	d_rowInd = (int const *)(mxGPUGetDataReadOnly(rowInd));
	d_colInd = (int const *)(mxGPUGetDataReadOnly(colInd));
	d_b = (double const *)(mxGPUGetDataReadOnly(b));

	/* Create a GPUArray to hold the result and get its underlying pointer. */
	mwSize X_num_dim = 1;
	mwSize X_dims[1]; // X_dmis[1] = {m};
	X_dims[0] = cols;
    X = mxGPUCreateGPUArray(X_num_dim, X_dims, mxGPUGetClassID(Aval), mxGPUGetComplexity(Aval), MX_GPU_DO_NOT_INITIALIZE);
    d_X = (double *)(mxGPUGetData(X));

	// ---------------------------------- CUBLAS initialization ---------------------------------------
	hipblasHandle_t cublas_handle;
	checkCublas(hipblasCreate(&cublas_handle));

	// ---------------------------------- CUSPARSE initialization -------------------------------------
	hipsparseHandle_t cusparse_handle = 0;  // po co przypisywać te zero ??
	hipsparseMatDescr_t descrA=0;
	//int *nnzPerRow, *csrRowPtrA, *csrColIndA;
	//double* csrValA;
	int *csrRowPtrA;
	int lda = rows;
	checkCusparseErrors(hipsparseCreate(&cusparse_handle));

	// ---------------------------------- CUFFT initialization ----------------------------------------
	hipfftHandle cufft_plan;
	checkCufft(hipfftPlan2d(&cufft_plan, n_sqrt, n_sqrt, HIPFFT_R2C));


	// ---------------------------------- rxk and x0 initialization -----------------------------------
	checkCudaErrors(hipMalloc((void**)&d_rxk, rows*sizeof(double)));
	checkCudaErrors(hipMemcpy(d_rxk, d_b, rows*sizeof(double), hipMemcpyDeviceToDevice)); // rxk = b
	if (nrhs < (required_args + 1)){
		checkCudaErrors(hipMalloc((void**)&d_x0, cols*sizeof(double)));
		checkCudaErrors(hipMemset(d_x0, 0, cols*sizeof(double)));
	}
	else{
		X0 = mxGPUCreateFromMxArray(prhs[required_args]);
		d_x0 = (double *)(mxGPUGetDataReadOnly(X0)); // czy na pewno read only??
	}

	// alokacja pamieci dla procedur cusparse
	//checkCudaErrors(hipMalloc((void**)&nnzPerRow, rows*sizeof(int)));

	// --------------------------------------- CUSPARSE CONVERSE DENSE TO CSR -------------------------------------------
	checkCusparseErrors(hipsparseCreateMatDescr(&descrA));
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	
	// as we get saprse coo format from matlab we no longer need compute nnzTotal, and nnzPerRow
	//checkCusparseErrors(hipsparseDnnz(cusparse_handle, dirA_row, rows, cols, descrA, d_A, lda, nnzPerRow, nnzTotal));
	
	checkCudaErrors(hipMalloc((void**)&csrRowPtrA, (rows+1)*sizeof(int)));
	//checkCudaErrors(hipMalloc((void**)&csrValA, (*nnzTotal)*sizeof(double)));
	//checkCudaErrors(hipMalloc((void**)&csrColIndA, (*nnzTotal)*sizeof(int)));
	
	// dalej w programie, dla obliczeń A*x0 będziemy potrzebować macierzy A w formacie CSR (compressed sparse row)
	//checkCusparseErrors(hipsparseDdense2csr(cusparse_handle, m, n, descrA, d_A, lda, nnzPerRow, csrValA, csrRowPtrA, csrColIndA));

	// --------- convert from coo sparse format (given already from matlab) to csr -------------------
	// a może skoro cols jest znacznie wieksze (262144) to może przechowywać w csc ??
	checkCusparseErrors(hipsparseXcoo2csr(cusparse_handle, d_rowInd, nnz, rows, csrRowPtrA, HIPSPARSE_INDEX_BASE_ZERO));


	// --------------------------------------- rxk CALCULATIONS --------------------------------------------------
	if (nrhs > required_args){ // jesli x0 jest podane jako argument, czyli A*x0 nie jest równe 0
		// rxk = b - A*x0, przy czym rxk juz jest rowne b, wiec robimy tylko, rxk - A*x0
		// Mnożenie A*x0, y = α ∗ op ( A ) ∗ x + β ∗ y
		// stare - checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, NON_TRANS, m, n, *nnzTotal, &negative, descrA, csrValA, csrRowPtrA, csrColIndA, d_x0, &positive, d_rxk));
		checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, NON_TRANS, rows, cols, nnz, &negative, descrA, d_Aval, csrRowPtrA, d_colInd, d_x0, &positive, d_rxk));
	}

	// -------------------------------------- V, W VECTORS CALCULATIONS ------------------------------------------
	/*
	1. Algorytm zwykłej redukcji
	2. Pomnożenie macierzy przez wektor jedynek
	3. Pomnożenie macierzy przez wektor jedynek,  - napisanie własnego kernela, gdzie nie ma tablicy wektora, tylko 1.0 z palca jest wpisany
	4. Jakieś próby z macierzą w formacie rzadkim (CSC, CSR)
	*/

	// ----------------- SPOSÓB NR 2 - MNOŻENIE MACIERZY PRZEZ WEKTOR JEDYNEK -------------------------------------
	// ----------------- SPOSÓB NR 2 - MNOŻENIE MACIERZY PRZEZ WEKTOR JEDYNEK -------------------------------------

	checkCudaErrors(hipMalloc((void**)&d_W, rows*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d_V, cols*sizeof(double)));

	HOST_ONES = (double*) malloc(ONES_SIZE*sizeof(double));
	initOnes(HOST_ONES, ONES_SIZE);
	// checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ONES_DEV), HOST_ONES, ONES_SIZE*sizeof(double))); -- constant device memory is not suitable for cusparse operations

	// d_A - macierz zwykla - cublas itp.
	// csrValA - macierz w formacie rzadkim - cusparse
	//checkCublas(hipblasDgemv(cublas_handle, hipblasOperation_t trans, m, n, const double * alpha, d_A, lda, ONES_DEV, 1, const double * beta, double * y, 1));
	
	double *d_ones;
	checkCudaErrors(hipMalloc((void**)&d_ones, ONES_SIZE*sizeof(double)));
	checkCudaErrors(hipMemcpy(d_ones, HOST_ONES, ONES_SIZE*sizeof(double), hipMemcpyHostToDevice));

	// y = α ∗ op ( A ) ∗ x + β ∗ y - csrmv
	// stare - checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, NON_TRANS, m, n, *nnzTotal, &positive, descrA, csrValA, csrRowPtrA, csrColIndA, ONES_DEV, &zero, d_W));
	// stare - checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, TRANS, m, n, *nnzTotal, &positive, descrA, csrValA, csrRowPtrA, csrColIndA, ONES_DEV, &zero, d_V));
	checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, NON_TRANS, rows, cols, nnz, &positive, descrA, d_Aval, csrRowPtrA, d_colInd, d_ones, &zero, d_W));
	checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, TRANS, rows, cols, nnz, &positive, descrA, d_Aval, csrRowPtrA, d_colInd, d_ones, &zero, d_V));

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	mexPrintf("Po mnozeniu macierzy A przez wektor jedynek\n");
	// a może strumieniowo ?
	int threads = 256;
	int blocks1D = 32;
	normalizeVectorSum<<<blocks1D, threads>>>(d_W, rows);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	
	// TODO konfiguracja kernela 
	normalizeVectorSum<<<cols/threads, threads>>>(d_V, cols);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	mexPrintf("po normalizacji normalizeVectorSum\n");
	// SPRAWDZIĆ JAK ZACHOWUJE SIĘ GPU PRZY DZIELENIU PRZEZ ZERO

	// tutaj możemy sprawdzić, czy wynik jest poprawny

	// Apj, Aip - wektory
    // do algorytmu redukcji możemy dołożyć 1./Aip
	// Apj = full(sum(abs(A),1)); % 1 - sumowanie po kolumnach
    // Aip = full(sum(abs(A),2)); % 2 - sumowanie po wierszach

	// W - rozmiar m, V - rozmiar n
	// W, V - wektory
    //  W = 1./Aip; przykladowe wartosci 0.0123
    // I = (W == Inf); 0 albo 1 - tam gdzie inf
    // W(I) = 0; zamienia tylko inf na zera
    // V = 1./Apj';
    // I = (V == Inf);
    // V(I) = 0;

	// W i V NIE SĄ wektorami rzadkimi, zdecydowana większość to elementy niezerowe
	// zakładamy, że m > n
	checkCudaErrors(hipMalloc((void**)&d_Wrxk, rows*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d_AW, cols*sizeof(double)));
	
	hipfftDoubleComplex *fft2_data;
	checkCudaErrors(hipMalloc((void**)&fft2_data, (cols/2+1)*sizeof(hipfftDoubleComplex))); // 2*sizeof(double)

	mexPrintf("przed petla while\n");
	int stop = 0;
	int iteration = 1;
	while(!stop){
		// rxk jest wektorem dense (gęsty, czyli nie rzadkim)
		// rxk = b - Ax, a wetkor b raczej nie jest rzadki, z 5400 elementow nieco ponad 3000 sa niezerowe
		if (casel == 1){
			// SART using constant value of lambda.
			// xk = xk + lambda*(V.*(A'*(W.*rxk)));
			// xk to x0

			/*
				1. v = W.*rxk
				2. z = A'*v
				3. xk + lambda*(V.*z)
				xk siedzi chyba w d_x0
			*/
			// d_AW =  W.*rxk, dlugosc m - chyba nie
			threads = 256;
			blocks1D = 32;
			elemByElem<<<blocks1D, threads>>>(rows, d_W, d_rxk, d_Wrxk);
			hipDeviceSynchronize();
			checkCudaErrors(hipGetLastError());
			//mexPrintf("po elemByElem\n");

		
			checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, TRANS, rows, cols, nnz, &positive, descrA, d_Aval, csrRowPtrA, d_colInd, d_Wrxk, &zero, d_AW));
			checkCudaErrors(hipGetLastError());
			//mexPrintf("po cusparse csrmv dot\n");			

			threads = 256;
			blocks1D = 32;
			saxdotpy<<<cols/threads, threads>>>(lambda, d_V, d_AW, cols, d_x0);

			hipDeviceSynchronize();
			checkCudaErrors(hipGetLastError());
			// jakies sprawdzenie bledów ??
		}
		else if(casel == 2){
			// SART using line search
			// TODO
		}
		else if(casel == 3){
			// SART using psi1 or psi2
			// TODO
		}

		// ----------------------- START REKONSTRUKCJA REC ----------------------------------------------------

		// reshape - bardzo ważne - trzeba przetransformować wektor na macierz
	    // U = reshape(xx,nn,nn); nn = sqrt(n)
	    // założenie jest takie, że nn jest liczbą całkowitą (n jest kwadratem liczby całkowitej)

		// fft2 jest obliczane dla liczb rzeczywistych

		//mexPrintf("przed fft2\n");
		hipfftExecD2Z(cufft_plan, d_x0, fft2_data);
		mexPrintf("po fft2\n");

		// FB - tu siedzi FFT2
		// fb = FB(:);
		// thresh = var(abs(fb))*median(abs(fb(2:end)))*max(10+k,10+K);%(K-k+1);
		// picks = find(abs(FB)>thresh);
		// B = FB(picks);
		// [UU,Out_RecPF] = RecPF(nn,nn,aTV,aL1,picks,B,2,opts,PsiT,Psi,range(U(:)),U);

		// ----------------------- KONIEC REKONSTRUKCJI REC ---------------------------------------------------

		// ------------------------------------ NEW RESIDUAL ------------------------------------------------
		// rxk = b - A*xk;
		// rxk - d_rxk, 
		// b - d_b
		// może trzeba zrobić, że wcześniej w rxk siedzi już b

		// TODO poniższa linika kopiowania pamięci do optymalizacji !!!
		checkCudaErrors(hipMemcpy(d_rxk, d_b, rows*sizeof(double), hipMemcpyDeviceToDevice));
		checkCusparseErrors(hipsparseDcsrmv(cusparse_handle, NON_TRANS, rows, cols, nnz, &negative, descrA, d_Aval, csrRowPtrA, d_colInd, d_x0, &positive, d_rxk));

		// stopping rule - OPAKOWAĆ TO W FUNKCJĘ
		//stop = stopping_rule();
		if (iteration >= K)
			stop = 1;
		iteration++;
	} //koniec pętli while


	// ----------------------------------------- WRAP RESULTS FOR MATLAB --------------------------------------
	checkCudaErrors(hipMemcpy(d_X, d_x0, cols*sizeof(double), hipMemcpyDeviceToDevice));
	/* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(X);	

   /* The mxGPUArray pointers are host-side structures that refer to device
    * data. These must be destroyed before leaving the MEX function.  */
    //mxGPUDestroyGPUArray(A);
	mxGPUDestroyGPUArray(b);
    mxGPUDestroyGPUArray(X);

	mxGPUDestroyGPUArray(Aval);
	mxGPUDestroyGPUArray(rowInd);
	mxGPUDestroyGPUArray(colInd);

	// ------------------------------ CLEANUP -----------------------------
	hipfftDestroy(cufft_plan);
	checkCublas(hipblasDestroy(cublas_handle));
	checkCusparseErrors(hipsparseDestroyMatDescr(descrA));
	checkCusparseErrors(hipsparseDestroy(cusparse_handle));
	free(HOST_ONES);
	//free(nnzTotal);
	checkCudaErrors(hipFree(d_ones));
	checkCudaErrors(hipFree(fft2_data));
	checkCudaErrors(hipFree(d_rxk));
	//checkCudaErrors(hipFree(nnzPerRow));
	checkCudaErrors(hipFree(d_Wrxk));
	checkCudaErrors(hipFree(d_AW));
	checkCudaErrors(hipFree(d_W));
	checkCudaErrors(hipFree(d_V));
	//checkCudaErrors(hipFree(csrValA));
	checkCudaErrors(hipFree(csrRowPtrA));
	//checkCudaErrors(hipFree(csrColIndA));

	if (nrhs < (required_args+1)){
		hipFree(d_x0);
	}
	else{
		mxGPUDestroyGPUArray(X0);
	}

	// THE END
}

void verifyRetrievedPointers(mxGPUArray const *Aval, mxGPUArray const *rowInd, mxGPUArray const *colInd, mxGPUArray const *b){
	// Verify that Aval, rowInd, colInd and b really are double array before extracting the pointer.
    if (mxGPUGetClassID(Aval) != mxDOUBLE_CLASS) {
		hipDeviceReset();
		const char * errMsg = "Invalid Input argument: Aval is not a double array";
        mexErrMsgIdAndTxt(errId, errMsg); // errMsg
    }
	if (mxGPUGetClassID(rowInd) != mxINT32_CLASS) {
		hipDeviceReset();
        mexErrMsgIdAndTxt(errId, "Invalid Input argument: rowInd is not a mxINT32_CLASS array, it is %d\n", mxGPUGetClassID(rowInd)); // errMsg
    }
	if (mxGPUGetClassID(colInd) != mxINT32_CLASS) {
		hipDeviceReset();
		const char * errMsg = "Invalid Input argument: colInd is not a mxINT32_CLASS array";
        mexErrMsgIdAndTxt(errId, errMsg);
    }
	if (mxGPUGetClassID(b) != mxDOUBLE_CLASS) {
		hipDeviceReset();
		const char * errMsg = "Invalid Input argument: b is not a double array";
        mexErrMsgIdAndTxt(errId, errMsg);
    }
}

void checkCublas(hipblasStatus_t status){
	//mexPrintf("cublas status %d\n", status);
	if (status != HIPBLAS_STATUS_SUCCESS){
		hipDeviceReset();
		mexErrMsgIdAndTxt("Cublas error ", "code error: %d\n", status);
	}
}

void verifyArguments(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){
	/* Throw an error if the input is not a GPU array. */
	// mexFunction arguments (Aval, rowInd, colInd, nnz, rows, cols, b, K)
	if (nrhs < 8){
		hipDeviceReset();
		mexErrMsgIdAndTxt(errId, "Invalid Input argument: nrhs < 8");
	}
	else if(!(mxIsGPUArray(prhs[0]))){
		hipDeviceReset();
        mexErrMsgIdAndTxt(errId, "Invalid Input argument: prhs[0] is not a GPU array");
	}
	else if(!(mxIsGPUArray(prhs[1]))){
		hipDeviceReset();
        mexErrMsgIdAndTxt(errId, "Invalid Input argument: prhs[1] is not a GPU array");
	}
	else if(!(mxIsGPUArray(prhs[2]))){
		hipDeviceReset();
        mexErrMsgIdAndTxt(errId, "Invalid Input argument: prhs[2] is not a GPU array");
	}
	else if(!(mxIsGPUArray(prhs[6]))){
		hipDeviceReset();
        mexErrMsgIdAndTxt(errId, "Invalid Input argument: prhs[6] is not a GPU array");
	}
}

void checkCufft(hipfftResult_t status){
	if (status != HIPFFT_SUCCESS) {
		hipDeviceReset();
		mexErrMsgIdAndTxt("cuFFT error ", "code error %d\n", status);
	}
}


void exitProgramWithErrorMessage(char * error_message){
	mexErrMsgIdAndTxt("%s\n", error_message);
}

void initOnes(double *p, int n){
	int i;
	for (i=0; i<n; i++){
		p[i] = 1.0;
	}
}

int stopping_rule(char * stoprule, int k, int kmax){
	
	if (strcmp(stoprule, "DP")){
		//TODO
	}
	else if(strcmp(stoprule, "ME")){

	}
	else if(strcmp(stoprule, "NC")){

	}
	else if(strcmp(stoprule, "NO")){
		// NO - no stopping rule
		if(k >= kmax)
			return 1;
		return 0;
	}
	return 0;
}

__global__ void normalizeVectorSum(double * v, int n){

	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if(index < n){
		if (v[index] != 0.0){
			v[index] = 1.0 / v[index];
		}
	}
}


__global__ void saxdotpy(double a, double * x, double *y, double n, double *z){

	// wykonuje operacje
	// z = z + a*(x.*y);

	// TODO
	// stride version

	int index = threadIdx.x + blockDim.x*blockIdx.x;

	if (index < n){
		z[index] += a*x[index]*y[index]; 
		//x[index] = a*y[index];
	}
}

__global__ void elemByElem(int n, double *x, double *y, double *z){

	int index = threadIdx.x + blockDim.x*blockIdx.x;

	if (index < n){
		z[index] = x[index]*y[index]; 
	}
}

/*compute sqrt root of complex c
	Newtow's method for computing sqrt
*/
__device__ __inline__ hipDoubleComplex sqrtComplex(hipDoubleComplex c){

	//Csub - subtract two double complex number: x - y
	//Cmul - multiplicate two double complex number: x*y

	hipDoubleComplex x = c;
	hipDoubleComplex real2 = make_hipDoubleComplex (2.0, 0.0);
	/*
	for(unsigned iter=0; iter<10; iter++){
		x = hipCsub(x,hipCdivf(hipCsub(hipCmul(x,x), c), hipCmul(real2,x))); //
	}*/

	//we can unroll the loop - czy na pewno??
	/*1*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*2*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*3*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*4*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*5*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*6*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*7*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*8*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*9*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));
	/*10*/ x = hipCsub(x,hipCdiv(hipCsub(hipCmul(x,x), c), hipCmul(real2,x)));

/*
	int iter;
	for(iter=0; iter<10; iter++){
		x = hipCsubf(x,hipCdivf(hipCsubf(hipCmulf(x,x), c), hipCmulf(real2,x))); //
	}
*/

	return x;
}

__global__ void absComplex(hipfftDoubleComplex * idata, double *odata, int n){
	/*
		Instead of completely eliminating the loop when parallelizing the computation, 
		a grid-stride loop approach is used here
	*/

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
		hipfftDoubleComplex c = idata[i];
		double x2 = c.x*c.x; // pow2
		double y2 = c.y*c.y; // pow2
		odata[i] = sqrt(x2+y2);
	}
}

static const char *_cublasGetErrorEnum(hipblasStatus_t status)
{
    switch (status)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "cublas_success";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "cublasNotInitialized";

		case HIPBLAS_STATUS_ALLOC_FAILED:
			return "cublasAllocFailed";

		case HIPBLAS_STATUS_INVALID_VALUE:
			return "cublasInvalidValue";

		case HIPBLAS_STATUS_ARCH_MISMATCH:
			return "cublasArchMismatch";

		case HIPBLAS_STATUS_MAPPING_ERROR:
			return "cublasMappingError";
		
		case HIPBLAS_STATUS_EXECUTION_FAILED:
			return "cublasExecutionFailed";

		case HIPBLAS_STATUS_INTERNAL_ERROR:
			return "cublasInternalError";

		case HIPBLAS_STATUS_NOT_SUPPORTED:
			return "cublaseNotSupported";

		case HIPBLAS_STATUS_UNKNOWN:
			return "cublasLicenseError";
	}
	   
	return "<unknown>";
}