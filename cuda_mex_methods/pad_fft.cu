#include "hip/hip_runtime.h"

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"
#include "hip/hip_runtime_api.h"
#include <hipfft/hipfft.h>

__global__ void fill_after_fft(hipfftDoubleComplex* v, int N, int n, double * real_fft);
__global__ void psf_from_fft(hipfftDoubleComplex* vn, hipfftDoubleComplex* vm, int N, int n, int M, int m, double * real_fft);

void check_cufft(hipfftResult status);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

	 /* Declare all variables.*/
    mxGPUArray *A;
    mxGPUArray *B;
    double *d_A;
    double *d_B;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs < 3) || !(mxIsGPUArray(prhs[0]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    A = mxGPUCopyFromMxArray (prhs[0]); // mxGPUCreateFromMxArray(prhs[0]);
	int N = mxGetScalar(prhs[1]);
	int M = mxGetScalar(prhs[2]);

    /*
     * Verify that A really is a double array before extracting the pointer.
     */
    if (mxGPUGetClassID(A) != mxDOUBLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_A = (double *)(mxGPUGetData(A));

    /* Create a GPUArray to hold the result and get its underlying pointer. */
	mwSize B_dims[2];
	B_dims[0] = M;
	B_dims[1] = N;
    B = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            B_dims,//mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (double *)(mxGPUGetData(B));
	// ----------------- END OF MEX FILE STARTING CONFIGURATION ----------------------------

	//  --------------- REC BREGMAN METHODS STARTS HERE -------------------------------------------
	hipfftHandle plan, plan1d_m;
	hipfftDoubleComplex *output, *fft1d_m;
	double prd[2] = {0.57, -0.57};
	double * input_n, *input_m;

	int n = (N/2)+1;
	//int m_fft = (M/2)+1;

	//---------------------- MEMORY ALLOCATION
	checkCudaErrors(hipMalloc((void**)&output, sizeof(hipfftDoubleComplex)*n));
	checkCudaErrors(hipMalloc((void**)&fft1d_m, sizeof(hipfftDoubleComplex)*m_fft);

	hipMalloc((void**)&input_n, sizeof(double)*N);
	hipMemset(input_n, 0, sizeof(double)*N);

	hipMalloc((void**)&input_m, sizeof(double)*M);
	hipMemset(input_m, 0, sizeof(double)*M);

	mexPrintf("po memset\n");
	// SET prd
	hipMemcpy(input_n, &prd[0], sizeof(double)*2, hipMemcpyHostToDevice);
	hipMemcpy(input_m, &prd[0], sizeof(double)*2, hipMemcpyHostToDevice);
	mexPrintf("po 0 i 1\n");

	// cufft plan initialization
	if (hipfftPlan1d(&plan, N, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS){	// N - number of input samples (length of input data)
		hipFree(output);
		mexErrMsgIdAndTxt(errId, "plan initialization failed, cufft error code\n");
	}
	if (hipfftPlan1d(&plan1d_m, M, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS){	// N - number of input samples (length of input data)
		hipFree(output);
		mexErrMsgIdAndTxt(errId, "plan initialization failed, cufft error code\n");
	}

	// exec
	if (hipfftExecD2Z(plan, input_n, output) != HIPFFT_SUCCESS){
		hipFree(output);
		hipfftDestroy(plan);		
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}
	if (hipfftExecD2Z(plan1d_m, input_m, fft1d_m) != HIPFFT_SUCCESS){
		hipFree(fft1d_m);
		hipfftDestroy(plan1d_m);		
		mexErrMsgIdAndTxt(errId, "cufft exec failed, cufft error code\n");
	}


	int threads = 256;
	int blocks = 32;
	fill_after_fft<<<threads, blocks>>>(output, N, n, d_B);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());		
	// END OF MY CODE

	/* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(B);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(B);

	hipfftDestroy(plan);
	hipfftDestroy(plan1d_m);

	hipFree(output);
	hipFree(input_n);
	hipFree(fft1d_m);
}

__global__ void fill_after_fft(hipfftDoubleComplex* v, int N, int n, double * real_fft){

	int index = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (index >= n && index < N){
		v[index] = v[N-index];
	}

	if (index < N){
		hipfftDoubleComplex a = v[index];
		//real_fft[index] = pow(a.x, 2.0) + pow(a.y, 2.0);
		v[index] = pow(a.x, 2.0) + pow(a.y, 2.0);
	}
}

__global__ void psf_from_fft(hipfftDoubleComplex* vn, hipfftDoubleComplex* vm, int N, int n, int M, int m, double * real_fft){

	int index = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (index >= n && index < N){
		v[index] = v[N-index];
	}

	if (index < N){
		hipfftDoubleComplex a = v[index];
		//real_fft[index] = pow(a.x, 2.0) + pow(a.y, 2.0);
		v[index] = pow(a.x, 2.0) + pow(a.y, 2.0);
	}

	// now comes part of merging to vectors into matrix
	// in this point we assume that vectors vm and vn are already normalized to pow
	__shared__ double[M] vm_shared;
	__shared__ double[N] vn_shared;

	vm_shared[some_m_index] = vm[some_m_index];
	vn_shared[some_n_index] = vn[some_n_index];
}


void check_cufft(hipfftResult status){
	
	char const * const errId = "parallel:gpu:mexGPUExample:CufftError";
	if (status != HIPFFT_SUCCESS){
		mexErrMsgIdAndTxt(errId, "cufft error code %d\n", status);
	}
}