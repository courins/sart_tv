
#include <hip/hip_runtime.h>
/*
	cuda implementation of RecPF algorithm based on matlab function version

	[U,Out] = RecPF(m,n,aTV,aL1,picks,B,TVtype,opts,PsiT,Psi,URange,uOrg) - deklaracja funkcji w Matlabie

	poni�ej przyk�adowe wywo�anie funkcji RecPF w matlabie ze skryptu sart_tv
	[UU,Out_RecPF] = RecPF(nn,nn,aTV,aL1,picks,B,2,opts,PsiT,Psi,range(U(:)),U);

	U - macierz
	U(:) - wektor, dodaje kolejno kolumny macierzy U
	range - dla wektora, zwraca r�nice mi�dzy najbardziej skrajnymi warto�ciami (najmniejsz� i najwi�ksz�)
			innymi s�owy, zwraca najwi�ksz� r�nic� mi�dzy dowolnymi dwoma elementami

	opis parametr�w:
	czy parametr jest skalarem, wektorem, macierz�, typ danych itp.

	m - nn - warto�� typu int
	n - nn - warto�� typu int
	aTV - sta�a double 
	aL1 - sta�a double
	picks - wektor zawieraj�cy indeksy, patrz: picks = find(abs(FB)>thresh);
	B - B = FB(picks); - tworzy wektor B z warto�ciami z macierzy FB odpowiadaj�cymi indeksom picks
	TVtype -- 2 (isotropic) or 1 (anisotropic) (w przyk�adzie warto�� 2)
	opts
	Psit - chyba funkcja
	Psi - chyba funkcja
	URange  - range(U(:))
	uOrg -- (optional) true image - macierz

	-----w sart_tv: ----------
	fb = FB(:);
	U = reshape(xx,nn,nn); 
    FB = fft2(U)/nn;%sqrt(n);
	thresh = var(abs(fb))*median(abs(fb(2:end)))*max(10+k,10+K);%(K-k+1);
    picks = find(abs(FB)>thresh);
	B = FB(picks);
	----------------------------

	tresh - ta warto�� jest obliczana w spos�b do�� skomplikowany
	
	picks, B oraz U, kt�re s� argumentami w wywo�aniu RecPF
	B = FB(picks); - tworzy wektor B z warto�ciami z macierzy FB odpowiadaj�cymi indeksom picks
*/

/*
algorytm do obliczania Denom2 = abs(psf2otf([prd,-prd],[m,n])).^2 + abs(psf2otf([prd;-prd],[m,n])).^2; 

abs(psf2otf([prd,-prd],[m,n])).^2 = abs(fft([prd,-prd], n)).^2 (gdzie fft daje tylko wiersz, kt�ry nale�y powieli�)
abs(psf2otf([prd;-prd],[m,n])).^2 = abs(fft([prd;-prd], m)).^2 (gdzie fft daje tylko kolumn�, kt�r� nale�y powieli�)

*/


