#include "hip/hip_runtime.h"
/*
	cuda implementation of RecPF algorithm based on matlab function version

	[U,Out] = RecPF(m,n,aTV,aL1,picks,B,TVtype,opts,PsiT,Psi,URange,uOrg) - deklaracja funkcji w Matlabie

	poni�ej przyk�adowe wywo�anie funkcji RecPF w matlabie ze skryptu sart_tv
	[UU,Out_RecPF] = RecPF(nn,nn,aTV,aL1,picks,B,2,opts,PsiT,Psi,range(U(:)),U);

	U - macierz
	U(:) - wektor, dodaje kolejno kolumny macierzy U
	range - dla wektora, zwraca r�nice mi�dzy najbardziej skrajnymi warto�ciami (najmniejsz� i najwi�ksz�)
			innymi s�owy, zwraca najwi�ksz� r�nic� mi�dzy dowolnymi dwoma elementami

	opis parametr�w:
	czy parametr jest skalarem, wektorem, macierz�, typ danych itp.

	m - nn - warto�� typu int
	n - nn - warto�� typu int
	aTV - sta�a double 
	aL1 - sta�a double
	picks - wektor zawieraj�cy indeksy, patrz: picks = find(abs(FB)>thresh);
	B - B = FB(picks); - tworzy wektor B z warto�ciami z macierzy FB odpowiadaj�cymi indeksom picks
	TVtype -- 2 (isotropic) or 1 (anisotropic) (w przyk�adzie warto�� 2)
	opts
	Psit - chyba funkcja
	Psi - chyba funkcja
	URange  - range(U(:))
	uOrg -- (optional) true image - macierz

	-----w sart_tv: ----------
	fb = FB(:);
	U = reshape(xx,nn,nn); 
    FB = fft2(U)/nn;%sqrt(n);
	thresh = var(abs(fb))*median(abs(fb(2:end)))*max(10+k,10+K);%(K-k+1);
    picks = find(abs(FB)>thresh);
	B = FB(picks);
	----------------------------

	tresh - ta warto�� jest obliczana w spos�b do�� skomplikowany
	
	picks, B oraz U, kt�re s� argumentami w wywo�aniu RecPF
	B = FB(picks); - tworzy wektor B z warto�ciami z macierzy FB odpowiadaj�cymi indeksom picks
*/



/*
U = zeros(m,n);     % initial U. 
                    % If changing to anything nonzeor, you must change the 
                    % initialization of Ux and Uy below
					*/

#define BATCH_1 1

int m = 5400, n = 2500;
int mn = m*n;
int normalize = 1;
int prd_fft_output_size = m*(n/2+1);

// ----------------------------- initializing part --------------------------------
double *U, *Numer1, *Denom1, *Denom2, *Denom2_work, *prd_fft2;
checkCudaErrors(hipMalloc((void**)&U, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&Numer1, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&Denom1, mn*sizeof(double)));

checkCudaErrors(hipMalloc((void**)&Denom2, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&Denom2_work, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&prd_fft2, (mn+1)*2*sizeof(double))); //mno�ymy razy 2 bo to dla zespolonych, powinno by� sizeof(hipDoubleComplex)

double *Ux, *Uy, *bx, *by;
checkCudaErrors(hipMalloc((void**)&Ux, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&Uy, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&bx, mn*sizeof(double)));
checkCudaErrors(hipMalloc((void**)&by, mn*sizeof(double)));

double *PsiTU, *Z, *d;
if (aL1 > 0){
	checkCudaErrors(hipMalloc((void**)&PsiTU, mn*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&Z, mn*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d, mn*sizeof(double)));
	
	// memset to zero
	checkCudaErrors(hipMemset(PsiTU, 0, mn*sizeof(double)));
	checkCudaErrors(hipMemset(Z, 0, mn*sizeof(double)));
	checkCudaErrors(hipMemset(d, 0, mn*sizeof(double)));
}

checkCudaErrors(hipMemset(U, 0, mn*sizeof(double)));
checkCudaErrors(hipMemset(Numer1, 0, mn*sizeof(double)));
checkCudaErrors(hipMemset(Denom1, 0, mn*sizeof(double)));
checkCudaErrors(hipMemset(Ux, 0, mn*sizeof(double)));
checkCudaErrors(hipMemset(Uy, 0, mn*sizeof(double)));
checkCudaErrors(hipMemset(bx, 0, mn*sizeof(double)));
checkCudaErrors(hipMemset(by, 0, mn*sizeof(double)));

if (normalize){

}

// prd = sqrt(aTV*beta);
double prd = sqrt(aTV*beta);

/*
algorytm do obliczania Denom2 = abs(psf2otf([prd,-prd],[m,n])).^2 + abs(psf2otf([prd;-prd],[m,n])).^2; 

abs(psf2otf([prd,-prd],[m,n])).^2 = abs(fft([prd,-prd], n)).^2 (gdzie fft daje tylko wiersz, kt�ry nale�y powieli�)
abs(psf2otf([prd;-prd],[m,n])).^2 = abs(fft([prd;-prd], m)).^2 (gdzie fft daje tylko kolumn�, kt�r� nale�y powieli�)

*/
// Denom2 = abs(psf2otf([prd,-prd],[m,n])).^2 + abs(psf2otf([prd;-prd],[m,n])).^2; % mozemy na potrzeby CUDA zastapic to wywolaniem: fft([prd, -prd], n) - kolumny s� takie same

hipfftHandle plan1D_n, plan1D_m;
hipfftComplex *output_2, *output_1;
hipfftReal * input_1, *input_2;

hipMalloc((void**)&output_1, sizeof(hipfftComplex)*(n/2+1)); // a mo�e BATCH = 2?
hipMalloc((void**)&output_2, sizeof(hipfftComplex)*(m/2+1));
hipMalloc((void**)&input_1, sizeof(double)*n);
hipMalloc((void**)&input_2, sizeof(double)*m);

// padding data wih zero [prd,-prd]
hipMemset(input_1, 0, sizeof(double)*n)
hipMemset(inptu_2, 0, sizeof(double)*m);
input_1[0] = prd; input_1[1] = -prd;
input_2[0] = prd; input_2[1] = -prd;

// cufft plan
hipfftPlan1d(&plan_n, n, CUFFT_RC2, BATCH_1);
hipfftPlan2d(plan1D_m, m, CUFFT_RC2, BATCH_1);

// ?? pytanie czy w przypadku tej transformaty mo�emy j� zrobi� w miejscu??
// hipfftResult hipfftExecC2R(hipfftHandle plan, hipfftComplex *idata, hipfftReal *odata);
cufftExecRC2(plan_n, input_1, output_1);
cufftExecRC2(plan_m, input_2, output_2);

hipFree(input_1);
hipFree(input_2);
hipFree(output_1);
hipFree(output_2);

// ------------------------------- MAIN LOOP ----------------------------------
int maxItr = opts.maxItr;
int i;
for(i=0; i<max_iter; i++){

	// ---------------- w naszej wersji domy�lnie TV_type jest 2 -------------
	if (TV_type == 1){
		/*
			% anisotropic TV
            Ux = Ux + bx; Uy = Uy + by;      % latest Ux and Uy are already calculated
            Wx = sign(Ux).* max(abs(Ux)-1/beta,0);
            Wy = sign(Uy).* max(abs(Uy)-1/beta,0);
		*/
	}
	else if(TV_type == 2){
		/*
			% isotropic TV
            [Wx, Wy] = Compute_Wx_Wy(Ux,Uy,bx,by,1/beta);
		*/
	}
	else{
		// error
	}

    //   Z-subprolem
    if (aL1 > 0){
        // PsiTU = PsiTU + d;
        // Z = sign(PsiTU).*max(abs(PsiTU)-1/beta,0);
    }
}

// ------------------------------- cleanup part --------------------------------
hipFree(U);
hipFree(Numer1);
hipFree(Denom1);
hipFree(Denom2);
hipFree(Ux);
hipFree(Uy);
hipFree(bx);
hipFree(by);

hipFree(prd_fft2);

if (aL1 > 0){
	hipFree(PsiTU);
	hipFree(Z);
	hipFree(d);
}


__global__ __device__ void fill_after_fft(hipfftComplex* v, int N, int n){

	int index = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (index > N && index < n){
		v[index] = v[N-index];
	}
}