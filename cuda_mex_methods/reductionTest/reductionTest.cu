#include <stdio.h>
#include <hip/hip_runtime.h>
#include "reduction.cu"

int main(){

	double * tab, *odata, *result;
	int N = 512;
	int size = N*N;
	float milliseconds = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocation memory
	hipMalloc((void**)&tab, size*sizeof(double));
	hipMalloc((void**)&odata, N*sizeof(double));
	hipMalloc((void**)&result, sizeof(double));

	int sm_size = sizeof(double)*N;

	// reduction 0
	hipEventRecord(start);

	reduce0<<<N, N, sm_size>>>(tab, odata);
	reduce0<<<1, N, sm_size>>>(odata, result);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("reduction 0: %f ms\n", milliseconds);

	// reduction 1
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	reduce1<<<N, N, sm_size>>>(tab, odata);
	reduce1<<<1, N, sm_size>>>(odata, result);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("reduction 1: %f ms\n", milliseconds);

	// reduction 2
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	reduce2<<<N, N, sm_size>>>(tab, odata);
	reduce2<<<1, N, sm_size>>>(odata, result);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("reduction 2: %f ms\n", milliseconds);

	// reduction 3
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	reduce3<<<N/2, N, sm_size>>>(tab, odata);
	reduce3<<<1, N/2, sm_size>>>(odata, result);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("reduction 3: %f ms\n", milliseconds);

	// reduction 4
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	reduce4<<<N/2, N, sm_size>>>(tab, odata);
	reduce4<<<1, N/2, sm_size>>>(odata, result);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("reduction 4: %f ms\n", milliseconds);

	// reduction 5
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	reduce5<512><<<N/2, N, sm_size>>>(tab, odata);
	reduce5<256><<<1, N/2, sm_size>>>(odata, result);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("reduction 5: %f ms\n", milliseconds);



	// free memory
	hipFree(tab);
	hipFree(odata);
	hipFree(result);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}